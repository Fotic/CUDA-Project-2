#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <cmath>

// To size tou pinaka
#define size 3

__global__ void dinami(int* a, int* b, int* c) {
	
	for (int i = 0; i < size; i++) {
		c[threadIdx.x * size + i] = pow((double)a[threadIdx.x * size + i] - b[threadIdx.x * size + i],3);
	}
	
}

int main(void) {
	int A[size][size];      //Dimiourgo tous pinakes A,B,C
	int B[size][size];
	int C[size][size];
	int* dev_a;      //Dimiourgoume device copies tou a,b,c (pointers)
	int* dev_b;		//gia na stiloume ta dedomena stin GPU
	int* dev_c;
	int i = 0, j = 0;

	// Gemizo tous Pinakes A & B
	for ( i = 0; i < size; i++) {
		for ( j = 0; j < size; j++) {
			A[i][j] = rand() % 10;
			B[i][j] = rand() % 10;
			printf("A[%i][%i]: %i ", i, j, A[i][j]);
			printf("B[%i][%i]: %i \n", i, j, B[i][j]);
		}
		printf("\n");
	}

	// Dilonoume to megethos tou pinaka pou 8a xriastoume
	int size_2d = size * size * sizeof(int);

	// Desmeuo mnimi sto sistima
	hipMalloc(&dev_a, size_2d);
	hipMalloc(&dev_b, size_2d);
	hipMalloc(&dev_c, size_2d);

	// Copy ton dedomenon stin mnimi tis GPU (meso pointers)
	hipMemcpy(dev_a, A, size_2d, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, B, size_2d, hipMemcpyHostToDevice);
	hipMemcpy(dev_c, C, size_2d, hipMemcpyHostToDevice);

	// Kalo tin kernel
	dinami << < size, size >> > (dev_a, dev_b, dev_c);

	// Travao to output piso stin CPU
	hipMemcpy(C, dev_c, size_2d, hipMemcpyDeviceToHost);

	// Ta emfanizo
	printf("\n");
	for (i = 0; i < size; i++) {
		for (j = 0; j < size; j++) {
			printf("C[%i][%i]: %i \n", i, j, C[i][j]);
		}
	}

	// Eleutherono tin mnimi
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}